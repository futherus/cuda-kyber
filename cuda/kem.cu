#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdint.h>
#include <string.h>
#include <assert.h>
#include <stdio.h>
#include "params.h"
#include "kem.h"
#include "indcpa.h"
#include "verify.h"
#include "symmetric.h"
#include "randombytes.h"

#include <hip/hip_runtime.h>

#define GPU_ASSERT(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*************************************************
* Name:        crypto_kem_keypair_derand
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes per thread)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes per thread)
*              - uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with 2*KYBER_SYMBYTES random bytes per thread)
**
* Returns 0 (success)
**************************************************/
__global__ void
crypto_kem_keypair_derand(uint8_t *pk,
                          uint8_t *sk,
                          const uint8_t *coins,
                          uint32_t keypair_count)
{
  const int tid = threadIdx.x;
  const int block_size = blockDim.x;
  const int bid = blockIdx.x;
  const int coins_offset = ( bid * block_size + tid ) * 2 * KYBER_SYMBYTES;
  const int sk_offset = ( bid * block_size + tid ) * KYBER_SECRETKEYBYTES;
  const int pk_offset = ( bid * block_size + tid ) * KYBER_PUBLICKEYBYTES;

  if ( bid * block_size + tid < keypair_count )
  {
    indcpa_keypair_derand(pk + pk_offset, sk + sk_offset, coins + coins_offset);
    memcpy(sk + sk_offset + KYBER_INDCPA_SECRETKEYBYTES, pk + pk_offset, KYBER_PUBLICKEYBYTES);
    hash_h(sk + sk_offset + KYBER_SECRETKEYBYTES - 2 * KYBER_SYMBYTES, pk + pk_offset, KYBER_PUBLICKEYBYTES);
    /* Value z for pseudo-random output on reject */
    memcpy(sk + sk_offset + KYBER_SECRETKEYBYTES - KYBER_SYMBYTES, coins + coins_offset + KYBER_SYMBYTES, KYBER_SYMBYTES);
  }
}

#define MAX_BLOCK_SIZE 1024

/*************************************************
* Name:        crypto_kem_keypair
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_keypair(uint8_t *pk,
                       uint8_t *sk,
                       uint32_t keypair_count)
{
  dim3 block_dim (1, 1, 1);
  dim3 grid_dim (1, 1, 1);

  if (keypair_count <= MAX_BLOCK_SIZE) {
    block_dim.x = keypair_count;
    grid_dim.x = 1;
  } else {
    block_dim.x = MAX_BLOCK_SIZE;
    grid_dim.x = ( keypair_count + MAX_BLOCK_SIZE - 1 ) / MAX_BLOCK_SIZE;
  }

  uint8_t coins[2*KYBER_SYMBYTES] = {
    0xcb, 0x12, 0x61, 0xa8, 0xcf, 0x85, 0xa4, 0x8b, 0x5d, 0x37, 0xc1, 0x00, 0xb6, 0xb0, 0x2c, 0xfb,
    0x1b, 0x84, 0x78, 0xc6, 0x2f, 0xe1, 0xc7, 0xd0, 0xe2, 0xcc, 0x0b, 0x48, 0xe7, 0xb7, 0xae, 0xfd,
    0x7f, 0xe1, 0xa8, 0x95, 0xdb, 0xd9, 0x28, 0x88, 0x12, 0xf2, 0x68, 0xc0, 0x84, 0x8e, 0xe0, 0xa6,
    0x1f, 0xe5, 0xd3, 0x21, 0xbb, 0xcf, 0x6d, 0x3c, 0x98, 0xb5, 0x35, 0xc4, 0x74, 0xae, 0x1a, 0xb0,
  };
  // randombytes(coins, 2*KYBER_SYMBYTES);

  uint8_t *d_pk = nullptr;
  uint8_t *d_sk = nullptr;
  uint8_t *d_coins = nullptr;

  hipMalloc( &d_pk, keypair_count * CRYPTO_PUBLICKEYBYTES);
  hipMalloc( &d_sk, keypair_count * CRYPTO_SECRETKEYBYTES);
  hipMalloc( &d_coins, keypair_count * 2 * KYBER_SYMBYTES);
  assert( d_pk && d_sk && d_coins);

  // TODO: remove for, instead init memory by one call of hipMemcpy
  for (int i = 0; i < keypair_count; i++) {
    GPU_ASSERT( hipMemcpy( d_coins + i * 2 * KYBER_SYMBYTES, coins, 2*KYBER_SYMBYTES, hipMemcpyHostToDevice) );
  }

  printf( "gridDim (%d %d %d), blockDim (%d %d %d)\n", grid_dim.x, grid_dim.y, grid_dim.z, block_dim.x, block_dim.y, block_dim.z);

  #if   (KYBER_K == 2)
    pqcrystals_kyber512_cuda_keypair_derand<<<grid_dim, block_dim>>>(d_pk, d_sk, d_coins, keypair_count);
  #elif (KYBER_K == 3)
    pqcrystals_kyber768_ref_keypair_derand<<<1, 1>>>(d_pk, d_sk, d_coins);
  #elif (KYBER_K == 4)
    pqcrystals_kyber1024_ref_keypair_derand<<<1, 1>>>(d_pk, d_sk, d_coins);
  #endif

  GPU_ASSERT( hipGetLastError() );

  GPU_ASSERT( hipMemcpy( pk, d_pk, keypair_count * CRYPTO_PUBLICKEYBYTES, hipMemcpyDeviceToHost) );
  GPU_ASSERT( hipMemcpy( sk, d_sk, keypair_count * CRYPTO_SECRETKEYBYTES, hipMemcpyDeviceToHost) );
  hipFree( d_pk);
  hipFree( d_sk);
  hipFree( d_coins);
  return 0;
}

#if 0
/*************************************************
* Name:        crypto_kem_enc_derand
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - const uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with KYBER_SYMBYTES random bytes)
**
* Returns 0 (success)
**************************************************/
int crypto_kem_enc_derand(uint8_t *ct,
                          uint8_t *ss,
                          const uint8_t *pk,
                          const uint8_t *coins)
{
  uint8_t buf[2*KYBER_SYMBYTES];
  /* Will contain key, coins */
  uint8_t kr[2*KYBER_SYMBYTES];

  memcpy(buf, coins, KYBER_SYMBYTES);

  /* Multitarget countermeasure for coins + contributory KEM */
  hash_h(buf+KYBER_SYMBYTES, pk, KYBER_PUBLICKEYBYTES);
  hash_g(kr, buf, 2*KYBER_SYMBYTES);

  /* coins are in kr+KYBER_SYMBYTES */
  indcpa_enc(ct, buf, pk, kr+KYBER_SYMBYTES);

  memcpy(ss,kr,KYBER_SYMBYTES);
  return 0;
}

/*************************************************
* Name:        crypto_kem_enc
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_enc(uint8_t *ct,
                   uint8_t *ss,
                   const uint8_t *pk)
{
  uint8_t coins[KYBER_SYMBYTES];
  randombytes(coins, KYBER_SYMBYTES);
  crypto_kem_enc_derand(ct, ss, pk, coins);
  return 0;
}

/*************************************************
* Name:        crypto_kem_dec
*
* Description: Generates shared secret for given
*              cipher text and private key
*
* Arguments:   - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *ct: pointer to input cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - const uint8_t *sk: pointer to input private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0.
*
* On failure, ss will contain a pseudo-random value.
**************************************************/
int crypto_kem_dec(uint8_t *ss,
                   const uint8_t *ct,
                   const uint8_t *sk)
{
  int fail;
  uint8_t buf[2*KYBER_SYMBYTES];
  /* Will contain key, coins */
  uint8_t kr[2*KYBER_SYMBYTES];
  uint8_t cmp[KYBER_CIPHERTEXTBYTES+KYBER_SYMBYTES];
  const uint8_t *pk = sk+KYBER_INDCPA_SECRETKEYBYTES;

  indcpa_dec(buf, ct, sk);

  /* Multitarget countermeasure for coins + contributory KEM */
  memcpy(buf+KYBER_SYMBYTES, sk+KYBER_SECRETKEYBYTES-2*KYBER_SYMBYTES, KYBER_SYMBYTES);
  hash_g(kr, buf, 2*KYBER_SYMBYTES);

  /* coins are in kr+KYBER_SYMBYTES */
  indcpa_enc(cmp, buf, pk, kr+KYBER_SYMBYTES);

  fail = verify(ct, cmp, KYBER_CIPHERTEXTBYTES);

  /* Compute rejection key */
  rkprf(ss,sk+KYBER_SECRETKEYBYTES-KYBER_SYMBYTES,ct);

  /* Copy true key to return buffer if fail is false */
  cmov(ss,kr,KYBER_SYMBYTES,!fail);

  return 0;
}

#endif

