#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdint.h>
#include <string.h>
#include <assert.h>
#include <stdio.h>
#include "params.h"
#include "kem.h"
#include "indcpa.h"
#include "verify.h"
#include "symmetric.h"
#include "randombytes.h"

#include <hip/hip_runtime.h>

#define GPU_ASSERT(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*************************************************
* Name:        crypto_kem_keypair_derand
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*              - uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with 2*KYBER_SYMBYTES random bytes)
**
* Returns 0 (success)
**************************************************/
__global__ void
crypto_kem_keypair_derand(uint8_t *pk,
                          uint8_t *sk,
                          const uint8_t *coins)
{
  indcpa_keypair_derand(pk, sk, coins);
  memcpy(sk+KYBER_INDCPA_SECRETKEYBYTES, pk, KYBER_PUBLICKEYBYTES);
  hash_h(sk+KYBER_SECRETKEYBYTES-2*KYBER_SYMBYTES, pk, KYBER_PUBLICKEYBYTES);
  /* Value z for pseudo-random output on reject */
  memcpy(sk+KYBER_SECRETKEYBYTES-KYBER_SYMBYTES, coins+KYBER_SYMBYTES, KYBER_SYMBYTES);
}

/*************************************************
* Name:        crypto_kem_keypair
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_keypair(uint8_t *pk,
                       uint8_t *sk)
{
  uint8_t coins[2*KYBER_SYMBYTES] = {
    0xcb, 0x12, 0x61, 0xa8, 0xcf, 0x85, 0xa4, 0x8b, 0x5d, 0x37, 0xc1, 0x00, 0xb6, 0xb0, 0x2c, 0xfb,
    0x1b, 0x84, 0x78, 0xc6, 0x2f, 0xe1, 0xc7, 0xd0, 0xe2, 0xcc, 0x0b, 0x48, 0xe7, 0xb7, 0xae, 0xfd,
    0x7f, 0xe1, 0xa8, 0x95, 0xdb, 0xd9, 0x28, 0x88, 0x12, 0xf2, 0x68, 0xc0, 0x84, 0x8e, 0xe0, 0xa6,
    0x1f, 0xe5, 0xd3, 0x21, 0xbb, 0xcf, 0x6d, 0x3c, 0x98, 0xb5, 0x35, 0xc4, 0x74, 0xae, 0x1a, 0xb0,
  };
  // randombytes(coins, 2*KYBER_SYMBYTES);

  uint8_t *d_pk = nullptr;
  uint8_t *d_sk = nullptr;
  uint8_t *d_coins = nullptr;
  hipMalloc( &d_pk, CRYPTO_PUBLICKEYBYTES);
  hipMalloc( &d_sk, CRYPTO_SECRETKEYBYTES);
  hipMalloc( &d_coins, 2*KYBER_SYMBYTES);
  assert( d_pk && d_sk && d_coins);

  GPU_ASSERT( hipMemcpy( d_coins, coins, 2*KYBER_SYMBYTES, hipMemcpyHostToDevice) );

  #if   (KYBER_K == 2)
    pqcrystals_kyber512_ref_keypair_derand<<<1, 1>>>(d_pk, d_sk, d_coins);
  #elif (KYBER_K == 3)
    pqcrystals_kyber768_ref_keypair_derand<<<1, 1>>>(d_pk, d_sk, d_coins);
  #elif (KYBER_K == 4)
    pqcrystals_kyber1024_ref_keypair_derand<<<1, 1>>>(d_pk, d_sk, d_coins);
  #endif

  GPU_ASSERT( hipGetLastError() );

  GPU_ASSERT( hipMemcpy( pk, d_pk, CRYPTO_PUBLICKEYBYTES, hipMemcpyDeviceToHost) );
  GPU_ASSERT( hipMemcpy( sk, d_sk, CRYPTO_SECRETKEYBYTES, hipMemcpyDeviceToHost) );
  return 0;
}

#if 0
/*************************************************
* Name:        crypto_kem_enc_derand
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - const uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with KYBER_SYMBYTES random bytes)
**
* Returns 0 (success)
**************************************************/
int crypto_kem_enc_derand(uint8_t *ct,
                          uint8_t *ss,
                          const uint8_t *pk,
                          const uint8_t *coins)
{
  uint8_t buf[2*KYBER_SYMBYTES];
  /* Will contain key, coins */
  uint8_t kr[2*KYBER_SYMBYTES];

  memcpy(buf, coins, KYBER_SYMBYTES);

  /* Multitarget countermeasure for coins + contributory KEM */
  hash_h(buf+KYBER_SYMBYTES, pk, KYBER_PUBLICKEYBYTES);
  hash_g(kr, buf, 2*KYBER_SYMBYTES);

  /* coins are in kr+KYBER_SYMBYTES */
  indcpa_enc(ct, buf, pk, kr+KYBER_SYMBYTES);

  memcpy(ss,kr,KYBER_SYMBYTES);
  return 0;
}

/*************************************************
* Name:        crypto_kem_enc
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_enc(uint8_t *ct,
                   uint8_t *ss,
                   const uint8_t *pk)
{
  uint8_t coins[KYBER_SYMBYTES];
  randombytes(coins, KYBER_SYMBYTES);
  crypto_kem_enc_derand(ct, ss, pk, coins);
  return 0;
}

/*************************************************
* Name:        crypto_kem_dec
*
* Description: Generates shared secret for given
*              cipher text and private key
*
* Arguments:   - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *ct: pointer to input cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - const uint8_t *sk: pointer to input private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0.
*
* On failure, ss will contain a pseudo-random value.
**************************************************/
int crypto_kem_dec(uint8_t *ss,
                   const uint8_t *ct,
                   const uint8_t *sk)
{
  int fail;
  uint8_t buf[2*KYBER_SYMBYTES];
  /* Will contain key, coins */
  uint8_t kr[2*KYBER_SYMBYTES];
  uint8_t cmp[KYBER_CIPHERTEXTBYTES+KYBER_SYMBYTES];
  const uint8_t *pk = sk+KYBER_INDCPA_SECRETKEYBYTES;

  indcpa_dec(buf, ct, sk);

  /* Multitarget countermeasure for coins + contributory KEM */
  memcpy(buf+KYBER_SYMBYTES, sk+KYBER_SECRETKEYBYTES-2*KYBER_SYMBYTES, KYBER_SYMBYTES);
  hash_g(kr, buf, 2*KYBER_SYMBYTES);

  /* coins are in kr+KYBER_SYMBYTES */
  indcpa_enc(cmp, buf, pk, kr+KYBER_SYMBYTES);

  fail = verify(ct, cmp, KYBER_CIPHERTEXTBYTES);

  /* Compute rejection key */
  rkprf(ss,sk+KYBER_SECRETKEYBYTES-KYBER_SYMBYTES,ct);

  /* Copy true key to return buffer if fail is false */
  cmov(ss,kr,KYBER_SYMBYTES,!fail);

  return 0;
}

#endif

